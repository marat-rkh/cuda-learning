#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <random>
#include <sstream>
#include <chrono>

using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::microseconds;

void vecAddCPU(float* A, float* B, float* C, int n) {
    auto start = high_resolution_clock::now();
    for (size_t i = 0; i < n; i++) {
        C[i] = A[i] + B[i];
    }
    auto end = high_resolution_clock::now();
    std::cout << "Vec add (CPU) took " << duration_cast<microseconds>(end - start).count() << " microseconds" << std::endl;
}

__global__
void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A, float* B, float* C, int n) {
    auto start = high_resolution_clock::now();
    float* A_d, * B_d, * C_d;
    int size = n * sizeof(float);
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
    auto mem_copy_to_gpu_end = high_resolution_clock::now();
    std::cout << "Memory copy to GPU took " << duration_cast<microseconds>(mem_copy_to_gpu_end - start).count() << " microseconds" << std::endl;
    vecAddKernel << < ceil(n / 256.0), 256 >> > (A_d, B_d, C_d, n);
    auto kernel_call_end = high_resolution_clock::now();
    std::cout << "Kernel call took " << duration_cast<microseconds>(kernel_call_end - mem_copy_to_gpu_end).count() << " microseconds" << std::endl;
    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    auto mem_copy_to_host_end = high_resolution_clock::now();
    std::cout << "Memory copy to host took " << duration_cast<microseconds>(mem_copy_to_host_end - kernel_call_end).count() << " microseconds" << std::endl;
    std::cout << "Vec add (GPU) took " << duration_cast<microseconds>(mem_copy_to_host_end - start).count() << " microseconds" << std::endl;
}

constexpr int N = 1000000;

void generateRandomInput() {
    std::ofstream file("in.txt");
    if (!file) {
        std::cerr << "Error: Cannot open file for writing!" << std::endl;
        return;
    }
    std::random_device rd;
    std::mt19937 gen(rd()); // Mersenne Twister PRNG
    std::uniform_real_distribution<float> dist(1, 1000);
    for (size_t i = 0; i < N; i++) {
        file << dist(gen) << " ";
    }
    file << "\n";
    for (size_t i = 0; i < N; i++) {
        file << dist(gen) << " ";
    }
    file.close();
}

void readInput(std::vector<float>& A, std::vector<float>& B) {
    std::ifstream file("in.txt");
    if (!file) {
        std::cerr << "Error: Cannot open file for reading!" << std::endl;
        return;
    }
    std::string line;
    float num;
    if (std::getline(file, line)) {
        std::istringstream iss(line);
        while (iss >> num) {
            A.push_back(num);
        }
    }
    if (std::getline(file, line)) {
        std::istringstream iss(line);
        while (iss >> num) {
            B.push_back(num);
        }
    }
    file.close();
}

int main() {
    generateRandomInput();

    std::vector<float> A, B;
    readInput(A, B);
    
    std::vector<float> C(A.size());
    vecAdd(A.data(), B.data(), C.data(), N);

    std::cout << "Vectors added. The first 10 values:\n";
    std::cout << "A: ";
    for (size_t i = 0; i < 10; i++) {
        std::cout << A[i] << " ";
    }
    std::cout << "\n";
    std::cout << "B: ";
    for (size_t i = 0; i < 10; i++) {
        std::cout << B[i] << " ";
    }
    std::cout << "\n";
    std::cout << "C: ";
    for (size_t i = 0; i < 10; i++) {
        std::cout << C[i] << " ";
    }
    std::cout << "\n\n";

    vecAddCPU(A.data(), B.data(), C.data(), N);
    std::cout << "\n";
}