#include "hip/hip_runtime.h"
__global__
void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A, float* B, float* C, int n) {
    float* A_d, * B_d, * C_d;
    int size = n * sizeof(float);
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
    vecAddKernel << < ceil(n / 256.0), 256 >> > (A_d, B_d, C_d, n);
    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main() {
    int N = 10000;
    float *A = (float *)malloc(N * sizeof(float));
    float *B = (float *)malloc(N * sizeof(float));
    float *C = (float *)malloc(N * sizeof(float));
    vecAdd(A, B, C, N);
    free(A);
    free(B);
    free(C);
}